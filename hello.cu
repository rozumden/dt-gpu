#include "hip/hip_runtime.h"
#include <stdio.h>
#include "init.h"

__global__ void mykernel(){
    printf("Hello from GPU!\n");
}

void hello(){
	hipSetDevice(0);
    CHECK_ERROR(hipGetLastError());
    mykernel<<<1,1>>>();
    CHECK_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    CHECK_ERROR(hipGetLastError());
}