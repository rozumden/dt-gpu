#include "hip/hip_runtime.h"
#include <stdio.h>
#include "init.cu"

__global__ void mykernel(){
    printf("Hello from GPU!\n");
}

void hello(int dev){
	hipSetDevice(dev);
    CHECK_ERROR(hipGetLastError());
    mykernel<<<1,1>>>();
    CHECK_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    CHECK_ERROR(hipGetLastError());
}