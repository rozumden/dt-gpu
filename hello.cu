#include "hip/hip_runtime.h"
#include <stdio.h>
#include "init.h"

__global__ void mykernel(){
    printf("Hello from GPU!\n");
}

void hello(int dev){
	hipSetDevice(dev);
    CHECK_ERROR(hipGetLastError());
    mykernel<<<1,1>>>();
    CHECK_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    CHECK_ERROR(hipGetLastError());
}