#include "hip/hip_runtime.h"
#include <stdio.h>
#include "init.h"

__global__ void mykernel(){
    printf("Hello from GPU!\n");
}

void hello(){
	hipSetDevice(0);
    mykernel<<<1,1>>>();
    hipDeviceSynchronize();
    CHECK_ERROR(hipGetLastError());
}