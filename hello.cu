
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(){
    printf("Hello from GPU!\n");
}

void hello(){
    mykernel<<<1,1>>>();
    hipDeviceSynchronize();
}