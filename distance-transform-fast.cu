#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <assert.h>
#include <time.h>
#include "init.h"

__global__ void computeCol(BYTE* src, int* out, int sizeRow, int sizeCol) {
	extern __shared__ int imgCol []; // allocates shared memory
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int untilPixel = min(x + sizeRow/blockDim.x, sizeCol);
	int row, rowi;
	int d;
	int value;
	for (row = threadIdx.x; row < sizeCol; row += blockDim.x) {
		imgCol[row] = src[row*sizeRow+y]; // copy column to shared memory
	}
	__syncthreads();
	for(row = x; row < untilPixel; row += blockDim.x) {
		value = imgCol[row];
		if(value != 0) {
			value = sizeRow*sizeRow + sizeCol*sizeCol;
			d = 1;
			for(rowi = 1; rowi < sizeCol - row; rowi++) { // scan 1
				if(imgCol[row + rowi] == 0)
					value = min(value, d);
				d += 1 + 2 * rowi;
				if(d > value) break;
			}
			d = 1;
			for(rowi = 1; rowi <= row; rowi++) { // scan 2
				if(imgCol[row - rowi] == 0)
					value = min(value, d);
				d += 1 + 2 * rowi;
				if(d > value) break;
			}
		}
		out[row * sizeRow + y] = value; 
	}
}

__global__ void computeRow(int* out, float* res, int sizeRow, int sizeCol) {
	extern __shared__ int imgRow[]; // allocates shared memory
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * sizeRow;
	int untilPixel = min(x + sizeRow/blockDim.x, sizeRow);
 	int col, coli;
 	int value;
 	int d;
 	for(col = threadIdx.x; col < sizeRow; col += blockDim.x) {
 		imgRow[col] = out[y + col]; // copy rows to shared memory
 	}
 	__syncthreads();
 	for(col = x; col < untilPixel; col += blockDim.x) {
 		value = imgRow[col];
 		if(value != 0) {
	 		d = 1;
			for(coli = 1; coli < sizeRow - col; coli++) { // scan 1
				value = min(value, imgRow[col + coli] + d);
				d += 1 + 2 * coli;
				if(d > value) break;
			}
	 		d = 1;
			for(coli = 1; coli <= col; coli++) { // scan 2
				value = min(value, imgRow[col - coli] + d);
				d += 1 + 2 * coli;
				if(d > value) break;
			}
		}
 		res[y + col] = sqrt((double)value);
	}
 }

void gpuDTfast(const BYTE *diffData, float *dtData, int w, int h) {
    int MAXTH = 1024;
    int ARRAY_SIZE = w*h;
    
    BYTE *devSrc;
    int *devTemp;
    float *devOut;
    hipMalloc((void **) &devSrc, ARRAY_SIZE * sizeof(BYTE));
    hipMalloc((void **) &devTemp, ARRAY_SIZE * sizeof(int));
    hipMalloc((void **) &devOut, ARRAY_SIZE * sizeof(float));
    hipMemcpy(devSrc, diffData, ARRAY_SIZE*sizeof(BYTE), hipMemcpyHostToDevice);

    int TH = MAXTH;
    if(h < TH) TH = h;
    int DH = (int) ceil(h/(float)TH);
    dim3 dimGrid(DH, w, 1);
	computeCol<<<dimGrid, TH, h*sizeof(int)>>>(devSrc, devTemp, w, h);
	hipDeviceSynchronize();
	CHECK_ERROR(hipGetLastError());

	int TW = MAXTH;
    if(w < TW) TW = w;
    int DW = (int) ceil(w/(float)TW);
    dim3 dimGridr(DW, h, 1);
	computeRow<<<dimGridr, TW, w*sizeof(int)>>>(devTemp, devOut, w, h);
	hipDeviceSynchronize();
	CHECK_ERROR(hipGetLastError());

    hipMemcpy(dtData, devOut, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost );
    CHECK_ERROR(hipGetLastError());

    hipFree(devSrc);
    hipFree(devTemp);
    hipFree(devOut);
}