#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <assert.h>
#include <time.h>
#include "init.h"

surface<void, 2> surfRef5;
hipArray* cuInArray5;

__constant__ float mask5[5][5];

float a5 = 1;
float b5 = 1.4;
float c5 = 2.1969;
float tmpMask5[][5] = { {-1,c5,-1,c5,-1}, {c5,b5,a5,b5,c5}, {-1,a5,0,a5,-1},{c5,b5,a5,b5,c5}, {-1,c5,-1,c5,-1}};

__global__ void calcLM_5x5(BYTE *dst, int w, int h){
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int ind = row * w + col;
    if(row < h && col < w) {
        float eps = 0;
        float data;
        surf2Dread(&data, surfRef5, col* 4, row);
        
        bool islm = false;

        if(data >= 1.5) {
            float data1;
            surf2Dread(&data1, surfRef5, (col-1) * 4, row-1, hipBoundaryModeClamp);
            if(data - data1 >= eps) {
            surf2Dread(&data1, surfRef5, (col-1) * 4, row+0, hipBoundaryModeClamp);
            if(data - data1 >= eps) {
            surf2Dread(&data1, surfRef5, (col-1) * 4, row+1, hipBoundaryModeClamp);
            if(data - data1 >= eps) {
            surf2Dread(&data1, surfRef5, (col+0) * 4, row-1, hipBoundaryModeClamp);
            if(data - data1 >= eps) {
            surf2Dread(&data1, surfRef5, (col+0) * 4, row+1, hipBoundaryModeClamp);
            if(data - data1 >= eps) {
            surf2Dread(&data1, surfRef5, (col+1) * 4, row-1, hipBoundaryModeClamp);
            if(data - data1 >= eps) {
            surf2Dread(&data1, surfRef5, (col+1) * 4, row+0, hipBoundaryModeClamp);
            if(data - data1 >= eps) {
            surf2Dread(&data1, surfRef5, (col+1) * 4, row+1, hipBoundaryModeClamp);
            if(data - data1 >= eps) islm = true;
            }}}}}}}
        } 
        if(islm) {
            dst[ind] = 255;
        } else {
            dst[ind] = 0;
        }
    }
}

__global__ void initDT_5x5(BYTE *src, int w, int h){
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int ind = row * w + col;

    if(row < h && col < w) {
        float data = 0.f;
        if(src[ind] != 0)
            data = (float)w*h;

        surf2Dwrite(data, surfRef5, col*4, row);
    }
}

__global__ void calcDT_5x5(int w, int h, int *done){
    __shared__ int found;
    bool written = true;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    bool mainthread = (threadIdx.x + threadIdx.y == 0);

    if(row < h && col < w) {
        if(mainthread) {
            written = false;
            atomicExch(&found, 1);
        }
        __syncthreads();

        int inf = 2147483647;
        float data;
        float eps = 0;
        surf2Dread(&data, surfRef5, col* 4, row);
        if(data > 0 || mainthread) {
            float newData, oldData;
            newData = data;
            oldData = data;
            while(found > 0) {
                if(mainthread) {
                    atomicExch(&found, 0);
                }
                __syncthreads();

                oldData = newData;
                newData = inf;
                
                surf2Dread(&data, surfRef5, (col-1) * 4, row-1, hipBoundaryModeClamp);
                data += mask5[-1+2][-1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col-1) * 4, row+0, hipBoundaryModeClamp);
                data += mask5[-1+2][0+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col-1) * 4, row+1, hipBoundaryModeClamp);
                data += mask5[-1+2][1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col+0) * 4, row-1, hipBoundaryModeClamp);
                data += mask5[0+2][-1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col+0) * 4, row+1, hipBoundaryModeClamp);
                data += mask5[0+2][1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col+1) * 4, row-1, hipBoundaryModeClamp);
                data += mask5[1+2][-1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col+1) * 4, row+0, hipBoundaryModeClamp);
                data += mask5[1+2][0+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col+1) * 4, row+1, hipBoundaryModeClamp);
                data += mask5[1+2][1+2];
                if(newData - data > eps) newData = data;

                /// for c's
                surf2Dread(&data, surfRef5, (col+2) * 4, row-1, hipBoundaryModeClamp);
                data += mask5[2+2][-1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col+2) * 4, row+1, hipBoundaryModeClamp);
                data += mask5[2+2][1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col-2) * 4, row-1, hipBoundaryModeClamp);
                data += mask5[-2+2][-1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col-2) * 4, row+1, hipBoundaryModeClamp);
                data += mask5[-2+2][1+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col-1) * 4, row-2, hipBoundaryModeClamp);
                data += mask5[-1+2][-2+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col+1) * 4, row-2, hipBoundaryModeClamp);
                data += mask5[1+2][-2+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col-1) * 4, row+2, hipBoundaryModeClamp);
                data += mask5[-1+2][2+2];
                if(newData - data > eps) newData = data;

                surf2Dread(&data, surfRef5, (col+1) * 4, row+2, hipBoundaryModeClamp);
                data += mask5[1+2][2+2];
                if(newData - data > eps) newData = data;

                if(newData < oldData) {
                    surf2Dwrite(newData, surfRef5, col * 4, row);
                    atomicExch(&found, 1);
                }

                __syncthreads();
                if(mainthread && found > 0 && !written) {
                    atomicExch(done, 0);
                    written = true;
                }
            }
        }


    }
}

void gpuDTLM_5x5(const BYTE *diffData, float *dtData, BYTE *lmData, int w, int h) {
    int TH = 32;
    dim3 dimBlock(TH,TH);
    int DW = (int) ceil(w/(float)TH);
    int DH = (int) ceil(h/(float)TH);
    dim3 dimGrid(DW,DH);

    int ARRAY_SIZE = w*h;

    BYTE *devSrc, *devLM;
    int *done;
    int doneCpu = 1;
    hipMalloc((void **) &done, sizeof(int));
    hipMalloc((void **) &devSrc, ARRAY_SIZE * sizeof(BYTE));
    hipMalloc((void **) &devLM, ARRAY_SIZE * sizeof(BYTE));
    hipMemcpy(devSrc, diffData, ARRAY_SIZE*sizeof(BYTE), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL( mask5), tmpMask5, sizeof(float)*5*5) ;

    // surface
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMallocArray(&cuInArray5, &channelDesc, w, h, hipArraySurfaceLoadStore);

    cudaBindSurfaceToArray(surfRef5, cuInArray5, channelDesc);
    initDT_5x5<<<dimGrid, dimBlock>>>(devSrc, w, h);
    hipDeviceSynchronize();
    int i = 0;
    while(true) {
        i++;
        hipMemset(done,1,sizeof(int));
        calcDT_5x5<<<dimGrid, dimBlock, sizeof(int) >>>(w,h,done);
        hipDeviceSynchronize();
        hipMemcpy(&doneCpu, done, sizeof(int), hipMemcpyDeviceToHost);
        if(doneCpu > 0) break;
    }
    hipDeviceSynchronize();
    calcLM_5x5<<<dimGrid, dimBlock>>>(devLM, w, h);
    hipMemcpyFromArray(dtData, cuInArray5, 0, 0, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy(lmData, devLM, ARRAY_SIZE * sizeof(BYTE), hipMemcpyDeviceToHost );
    printf("Finished after %d iterations\n", i);
    CHECK_ERROR(hipGetLastError());
    
    hipFreeArray(cuInArray5);
    hipFree(devSrc);
    hipFree(devLM);
    hipFree(done);
}
