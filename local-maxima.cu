#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "stdlib.h"
#include <cmath>
#include "init.h"

/// Texture reference 
texture<float, hipTextureType2D, hipReadModeElementType> texRef;

/// Device array binded to texture
hipArray *cuArray;


/// Compute local maxima using texture memory.
/**
  Compute local maxima using texture memory.

  \param[out] dst  Output 8bit matrix with positive numbers (here 255) indicating local maxima.
  \param[in] w     Image widht
  \param[in] h     Image height
  \return void
*/
__global__ void findLM(BYTE *dst, int w, int h){
    double col = blockIdx.x*blockDim.x + threadIdx.x;
    double row = blockIdx.y*blockDim.y + threadIdx.y;
    int ind = row * w + col;
    float eps = 0;
    if(row < h && col < w) {
        row += 0.5;
        col += 0.5;
        row /= h;
        col /= w;
        double row1 = 1.f/h;
        double col1 = 1.f/w;

        float inp = tex2D(texRef, col, row);
        if(inp >= 1.5 &&
           inp - tex2D(texRef, col-col1, row) >= eps &&
           inp - tex2D(texRef, col-col1, row-row1) >= eps  &&
           inp - tex2D(texRef, col-col1, row+row1) >= eps &&
           inp - tex2D(texRef, col+col1, row) >= eps &&
           inp - tex2D(texRef, col+col1, row-row1) >= eps &&
           inp - tex2D(texRef, col+col1, row+row1) >= eps &&
           inp - tex2D(texRef, col, row-row1) >= eps &&
           inp - tex2D(texRef, col, row+row1) >= eps)
        {
            dst[ind] = 255;
        } else {
            dst[ind] = 0;
        }
              
    }
}

/// Compute local maxima from distance transform using texture memory.
/**
  Compute local maxima using texture memory.
  
  \param[in] src   Input float matrix with Euclidean distance transform values.
  \param[out] dst  Output 8bit matrix with positive numbers (here 255) indicating local maxima.
  \param[in] w     Image widht
  \param[in] h     Image height
  \return void
*/
void gpuLocalMaxima(const float *src, BYTE *dst, int w, int h){
    /// number of threads per blocks in one dimention
    int TH = 32;
    dim3 dimBlock(TH,TH);
    int DW = (int) ceil(w/(float)TH);
    int DH = (int) ceil(h/(float)TH);
    dim3 dimGrid(DW,DH);

    int ARRAY_SIZE = w*h;
    BYTE* devDst;

    /// init textures
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMallocArray(&cuArray, &channelDesc, w, h);
    hipMemcpyToArray(cuArray, 0, 0, src, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);

    texRef.addressMode[0] = hipAddressModeClamp;
    texRef.addressMode[1] = hipAddressModeClamp;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = true;
    hipBindTextureToArray(texRef, cuArray, channelDesc);
    //////

    hipMalloc((void **) &devDst, ARRAY_SIZE * sizeof(BYTE));

    /// main kernel
    findLM<<<dimGrid, dimBlock>>>(devDst,w,h);
    CHECK_ERROR(hipGetLastError());

    hipDeviceSynchronize();

    hipMemcpy(dst, devDst, ARRAY_SIZE*sizeof(BYTE), hipMemcpyDeviceToHost);

    hipFree(devDst);
    hipUnbindTexture(texRef);
    hipFreeArray(cuArray);
}

