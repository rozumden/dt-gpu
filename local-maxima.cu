#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "stdlib.h"
#include <cmath>
#include "init.h"

texture<float, hipTextureType2D, hipReadModeElementType> texRef;
hipArray *cuArray;

__global__ void findLM(BYTE *dst, int w, int h){
    double col = blockIdx.x*blockDim.x + threadIdx.x;
    double row = blockIdx.y*blockDim.y + threadIdx.y;
    int ind = row * w + col;
    float eps = 0;
    if(row < h && col < w) {
        row += 0.5;
        col += 0.5;
        row /= h;
        col /= w;
        double row1 = 1.f/h;
        double col1 = 1.f/w;

        float inp = tex2D(texRef, col, row);
        if(inp >= 1.5 &&
           inp - tex2D(texRef, col-col1, row) >= eps &&
           inp - tex2D(texRef, col-col1, row-row1) >= eps  &&
           inp - tex2D(texRef, col-col1, row+row1) >= eps &&
           inp - tex2D(texRef, col+col1, row) >= eps &&
           inp - tex2D(texRef, col+col1, row-row1) >= eps &&
           inp - tex2D(texRef, col+col1, row+row1) >= eps &&
           inp - tex2D(texRef, col, row-row1) >= eps &&
           inp - tex2D(texRef, col, row+row1) >= eps)
        {
            dst[ind] = 255;
        } else {
            dst[ind] = 0;
        }
              
    }
}

void gpuLocalMaxima(const float *src, BYTE *dst, int w, int h){
    int TH = 32;
    dim3 dimBlock(TH,TH);
    int DW = (int) ceil(w/(float)TH);
    int DH = (int) ceil(h/(float)TH);
    dim3 dimGrid(DW,DH);

    int ARRAY_SIZE = w*h;
    BYTE* devDst;

    /// textures
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMallocArray(&cuArray, &channelDesc, w, h);
    hipMemcpyToArray(cuArray, 0, 0, src, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);

    texRef.addressMode[0] = hipAddressModeClamp;
    texRef.addressMode[1] = hipAddressModeClamp;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = true;
    hipBindTextureToArray(texRef, cuArray, channelDesc);
    //////

    hipMalloc((void **) &devDst, ARRAY_SIZE * sizeof(BYTE));

    findLM<<<dimGrid, dimBlock>>>(devDst,w,h);
    CHECK_ERROR(hipGetLastError());

    hipDeviceSynchronize();

    hipMemcpy(dst, devDst, ARRAY_SIZE*sizeof(BYTE), hipMemcpyDeviceToHost);

    hipFree(devDst);
    hipUnbindTexture(texRef);
    hipFreeArray(cuArray);
}

